#include "hip/hip_runtime.h"
// rtsine.cpp STK tutorial program
texture<float, 1, hipReadModeElementType> d_potencial_tex;

#include <signal.h>
#include <assert.h>
#include "RtMidi.h"
#include "RtWvOut.h"
#include "RtWvIn.h"

#include "lib/avancem2.cu"
#include "lib/AOAUsbConnectionManager.h"

using namespace stk;

#define USB_IN 0x81    // In Port
#define USB_OUT 0x07   // Out Port

#define USB_VID 0x18D1 // Google Inc.
#define USB_PID 0x4EE7 // Pixel 3 debug

#define USB_ACCESSORY_PID 0x2D01
#define USB_ACCESSORY_PID_ALT 0x2D00

/* Comentar y descomentar para elegir lo que se usa, MIDI o USB */
// #define USE_MIDI
#define USE_USB

bool debugMode;
bool imprimir;

void initPresses();
void deInitPresses();

template <class T>
void init(T *data, int size, T def) {
  for (int i = 0; i < size; ++i) data[i] = def;
}

void ceroInit(float *data, int size) { init<float>(data, size, 0.); }

void gaussiana(float *data, int nodos, int cuerdas, float valor, float centro,
               float expo, int *nodoscuer) {
  float cosa = 0;
  float maximo;
  for (int i = 0; i < cuerdas; ++i) {
    int sentro = centro * nodoscuer[i];
    maximo = valor - valor * 0.01 * i;
    for (int j = 0; j < nodos; ++j) {
      cosa = -expo * ((j - sentro) * (j - sentro));
      data[j + i * nodos] = maximo * exp(cosa);
    }
  }
}

void una_gaussiana(float *data, int nodos, int cuerda, float valor,
                   float centro, float expo, int *nodos_cuer) {
  float cosa = 0;
  float maximo = valor;
  int i = cuerda;
  int sentro = centro * nodos_cuer[i];
  for (int j = 0; j < nodos_cuer[i]; j++) {
    cosa = -expo * ((j - sentro) * (j - sentro));
    data[j + i * nodos] = maximo * exp(cosa);
  }
}

void friccion_una_punta(StateManager* var, const float *fricc, const int *dedo, int cuerda_tocada) {
  if (debugMode) printf("Dedo  %d  \n", dedo[cuerda_tocada]);
  int j = cuerda_tocada;
  int cantNodos = var->state()->getcuerdas(cuerda_tocada, "nodos");
  int dedo_aca = dedo[cuerda_tocada];
  float orden = var->state()->getcuerdas(cuerda_tocada, "anchoPuntas");

  for (int i = 0; i < cantNodos; ++i) {
    float factor = 0.5 - fricc[j];
    if (i <= dedo[cuerda_tocada]) {
      factor =
        (var->state()->getcuerdas(j, "maxFriccionEnPunta") - fricc[cuerda_tocada]) * (exp(-orden * ((i - dedo_aca) * (i - dedo_aca))) +
                            exp(-orden * (i - cantNodos) * (i - cantNodos)));
    }

    var->state()->setfriccionSinDedo(i + j * var->state()->getcantMaximaNodos(), factor + fricc[j]);

    if (fricc[i + j * var->state()->getcantMaximaNodos()] > 1.)
      printf("Friccion mayor a uno %e  \n",  var->state()->getfriccionSinDedo(i + j * var->state()->getcantMaximaNodos()));
  }
}

void imprimir_parametros(int ncuerdas, int nodos_max, vector<float> const& masaPorNodo, vector<float> const& fricc,
                         float *fuerza, int *mic, int *nodoscuer) {
	if (imprimir) {
		FILE *parametrosout;
		parametrosout = fopen("salida/parmout.dat", "w");

		for (int i = 0; i < ncuerdas; i++) {
		    for (int j = 0; j < nodos_max; j++) {
		      int nodo = j + i * nodos_max;
		      fprintf(parametrosout, "%e %e %e \n", masaPorNodo[nodo], fuerza[nodo],
		              fricc[nodo]);
		    }
		 }
		 fclose(parametrosout);
	}
}

template <class T>
T* cudaMallocAndCopy(const vector<T> &vect) {
  T* cuda_pointer;
  hipMalloc((void **)&cuda_pointer, vect.size() * sizeof(T));
  hipMemcpy(cuda_pointer, vect.data(), vect.size() * sizeof(T),
             hipMemcpyHostToDevice);
  return cuda_pointer;
}

template <class T>
T* cudaFreeMallocAndCopy(const vector<T> &vect, T* cuda_pointer) {
  hipFree(cuda_pointer);
  return cudaMallocAndCopy<T>(vect);
}

void iim2(StateManager *var) {
  debugMode = var->state()->getdebugMode();
  imprimir = var->state()->getimprimir();
  // Set the global sample rate before creating class instances.
  // Attempt to instantiate MIDI output class.

  /*************************
   *  Inicializo variables *
   *************************/

  float volumen = 1.0 / 1700.0;
  int cuerda_tocada = 0;
  bool de_a_una = 0;
  float palanca = 1;
  bool retoque = 0;
  bool pedal = true;
  float intensidad = 1e-4;
  float expo = 2.6;

  unsigned int size_A = var->state()->getcantMaximaNodos() * var->state()->getcantCuerdas();
  unsigned int mem_size_A = sizeof(float) * size_A;

  unsigned int buffer_salida =
      sizeof(float) * var->state()->getnbufferii() * var->state()->getcantCuerdas() * 2;
  unsigned int buffer_entrada =
      sizeof(float) * var->state()->getnbufferii() * var->state()->getcanalesEntrada() * 3;

  unsigned int bcur = sizeof(bool) * var->state()->getcantCuerdas();

  // Imprimo informacion en archivo (en caso de debug activo)
  FILE *xyz;
  FILE *out;
  if (imprimir) {
    xyz = fopen("salida/coord.xyz", "w");
    out = fopen("salida/salida.ascii", "w");

    fprintf(out, " ## 'rate'= %d\n", var->state()->getsamplerate());
    fprintf(out, " ## 'tracks'= 2\n");
    fprintf(out, "## 'bits'= 24\n");
    fprintf(out, "## 'length'= %d\n", var->state()->getnbufferii() * var->state()->getnFrames());
    fprintf(out, "## 'Date'='2013-11-09'\n");
    fprintf(out, "## 'Software'='nanosampler'\n");
  }

  // Reservo memoria local necesaria
  float *X = (float *)malloc(mem_size_A);
  float *V = (float *)malloc(mem_size_A);
  float *Fext = (float *)malloc(mem_size_A);
  float *salida = (float *)malloc(buffer_salida);
  float *entrada = (float *)malloc(buffer_entrada);
  bool *tococ = (bool *)malloc(bcur);
  bool *activa = (bool *)malloc(bcur);
  bool *freno = (bool *)malloc(bcur);
  int *caca = (int *)malloc(var->state()->getcantCuerdas() * sizeof(int));
  int *dedo = (int *)malloc(var->state()->getcantCuerdas() * sizeof(int));
  int *dedov = (int *)malloc(var->state()->getcantCuerdas() * sizeof(int));
  int *mic = (int *)malloc(2 * var->state()->getcantCuerdas() * sizeof(int));
  int *nodoscuer = (int *)malloc(var->state()->getcantCuerdas() * sizeof(int));
  int *escribe = (int *)malloc(var->state()->getcantCuerdas() * sizeof(int));
  int *fuerzavieja = (int *)malloc(var->state()->getcantCuerdas() * sizeof(int));
  float *fricc = (float *)malloc(var->state()->getcantCuerdas() * sizeof(float));
  float *paneo = (float *)malloc(var->state()->getcantCuerdas() * sizeof(float));
  float *volcuer = (float *)malloc(var->state()->getcantCuerdas() * sizeof(float));
  float *pertu = (float *)malloc(512 * sizeof(float));
  float *pertup = (float *)malloc(512 * sizeof(float));
  float *pertud = (float *)malloc(512 * sizeof(float));
  float *potencial = (float *)malloc(var->state()->getnpot() * sizeof(float));
  float *d_X;
  float *d_V;
  float *d_Fr;
  float *d_Frd;
  float *d_M;
  float *d_Fext;
  float *d_xmin;
  float *d_salida;
  float *d_entrada;
  bool *d_tococ;
  bool *d_activa;
  bool *d_freno;
  int *d_dedo;
  int *d_dedov;
  int *d_mic;
  int *d_nodoscuer;
  int *d_escribe;
  float *d_pertu;
  float *d_pertup;
  float *d_pertud;
  float *d_potencial;

  // 8. allocate device memory
  hipMalloc((void **)&d_X, mem_size_A);
  hipMalloc((void **)&d_V, mem_size_A);
  hipMalloc((void **)&d_Fext, mem_size_A);
  hipMalloc((void **)&d_salida, buffer_salida);
  hipMalloc((void **)&d_entrada, buffer_entrada);
  hipMalloc((void **)&d_tococ, bcur);
  hipMalloc((void **)&d_activa, bcur);
  hipMalloc((void **)&d_freno, bcur);
  hipMalloc((void **)&d_dedo, var->state()->getcantCuerdas() * sizeof(int));
  hipMalloc((void **)&d_dedov, var->state()->getcantCuerdas() * sizeof(int));
  hipMalloc((void **)&d_mic, 2 * var->state()->getcantCuerdas() * sizeof(int));
  hipMalloc((void **)&d_nodoscuer, var->state()->getcantCuerdas() * sizeof(int));
  hipMalloc((void **)&d_escribe, var->state()->getcantCuerdas() * sizeof(int));
  hipMalloc((void **)&d_pertud, 512 * sizeof(float));
  hipMalloc((void **)&d_pertup, 512 * sizeof(float));
  hipMalloc((void **)&d_potencial, var->state()->getnpot() * sizeof(float));

  hipArray *tex;
  hipChannelFormatDesc channel = hipCreateChannelDesc<float>();
  hipMallocArray(&tex, &channel, var->state()->getnpot(), 1, hipArrayDefault);

  // 2. initialize host memory
  ceroInit(X, size_A);
  ceroInit(V, size_A);
  ceroInit(salida, var->state()->getnbufferii() * var->state()->getcantCuerdas() * 2);

  int nodosreales = 0;
  for (int ii = 0; ii < var->state()->getcantCuerdas(); ++ii) {
    caca[ii] = 0;
    fuerzavieja[ii] = 0;
    dedo[ii] = var->state()->getcantMaximaNodos();
    dedov[ii] = 0;
    fricc[ii] = var->state()->getcuerdas(ii, "friccion");
    nodoscuer[ii] = var->state()->getcuerdas(ii, "nodos");
    nodosreales += nodoscuer[ii];
    activa[ii] = 0;
    mic[2 * ii] = int((float)nodoscuer[ii] * 0.1);
    mic[2 * ii + 1] = int((float)nodoscuer[ii] * 0.3);
    paneo[ii] = 0.5;
    volcuer[ii] = 3.0f;
  }

  for (int ii = 0; ii < var->state()->getnpot(); ++ii) {
    float iii = (float)ii - ((float)var->state()->getnpot()) / 2.0f;

    float deltax = iii / 10.0f;
    float dx2 = deltax * deltax;
    float d2tot = dx2 + var->state()->getdistanciaEntreNodos();
    float d12 = sqrt(d2tot);
    potencial[ii] =
        (d12 - var->state()->getdistanciaEquilibrioResorte()) / (d12)*deltax -
        deltax;  // << Esta es la cuenta rememorando algo 2D con delta Y siempre
                 // igual
  }

  hipMemcpyToArray(tex, 0, 0, potencial, 2048 * sizeof(float),
                    hipMemcpyHostToDevice);
  hipBindTextureToArray(d_potencial_tex, tex, channel);

  d_potencial_tex.normalized = false;
  d_potencial_tex.filterMode = hipFilterModeLinear;
  d_potencial_tex.addressMode[0] = hipAddressModeBorder;

  for (int ii = 0; ii < var->state()->getnpot() && debugMode && imprimir; ++ii) {
    printf(" NN %e   \n", potencial[ii]);
  }

  gaussiana(Fext, var->state()->getcantMaximaNodos(), var->state()->getcantCuerdas(), 2.E-2,
            var->state()->getcentro(), 0.2, nodoscuer);

  d_M = cudaMallocAndCopy<float>(var->state()->getmasaPorNodo());
  d_Fr = cudaMallocAndCopy<float>(var->state()->getfriccionSinDedo());
  d_Frd = cudaMallocAndCopy<float>(var->state()->getfriccionConDedo());
  d_xmin = cudaMallocAndCopy<float>(var->state()->getminimosYtrastes());

  if (debugMode && imprimir) {
    for (int nn = 0; nn < var->state()->getcantCuerdas() && debugMode; nn++) {
      for (int nj = 0; nj < var->state()->getcantMaximaNodos(); nj++) {
        printf("El nodo %d de la cuerda %d tiene minimo %e \n", nj, nn,
        		var->state()->getminimosYtrastes(nj + var->state()->getcantMaximaNodos() * nn));
      }
    }
  }

  hipMemcpy(d_X, X, mem_size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_V, V, mem_size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_salida, salida, buffer_salida, hipMemcpyHostToDevice);

  hipMemcpy(d_Fext, Fext, mem_size_A, hipMemcpyHostToDevice);
  dim3 threads(var->state()->getcantMaximaNodos(), 1, 1);
  dim3 grid(var->state()->getcantCuerdas(), 1, 1);
  dim3 threads_caja(1024);
  dim3 grid_caja(var->state()->getnbufferii());

  // levanto el input de perturvación

  FILE *pertp;
  FILE *pertd;
  const char *mode = "r";

  pertd = fopen("entrada/dedo.dat", mode);
  pertp = fopen("entrada/pua.dat", mode);

  if (pertd == NULL || pertp == NULL) {
    fprintf(stderr, "Can't open input file in.list!\n");
    exit(1);
  }

  for (int i = 0; i < 512; i++) {
    pertud[i] = 0.;
    pertup[i] = 0.;
  }

  int nn = 0;
  while (fscanf(pertp, "%d %E", &nn, &pertup[nn]) != EOF);
  while (fscanf(pertd, "%d %E", &nn, &pertud[nn]) != EOF);

  hipMemcpy(d_pertud, pertud, 512 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_pertup, pertup, 512 * sizeof(float), hipMemcpyHostToDevice);
  d_pertu = d_pertud;
  float random;

  hipMemcpy(d_dedo, dedo, var->state()->getcantCuerdas() * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_dedov, dedov, var->state()->getcantCuerdas() * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_mic, mic, 2 * var->state()->getcantCuerdas() * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_nodoscuer, nodoscuer, var->state()->getcantCuerdas() * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_activa, activa, bcur, hipMemcpyHostToDevice);

  init<bool>(freno, var->state()->getcantCuerdas(), false);

  imprimir_parametros(var->state()->getcantCuerdas(), var->state()->getcantMaximaNodos(),
                      var->state()->getmasaPorNodo(), var->state()->getfriccionSinDedo(), Fext,
                      mic, nodoscuer);

  init<bool>(tococ, var->state()->getcantCuerdas(), false);

  for (int cuerda_t = 0; cuerda_t < var->state()->getcantCuerdas();
       cuerda_t++) {
    intensidad = var->state()->getescalaIntensidad() * 600;
    una_gaussiana(Fext, var->state()->getcantMaximaNodos(), cuerda_t, intensidad,
                  var->state()->getcentro(), expo, nodoscuer);
  }

  hipMemcpy(d_Fext, Fext, mem_size_A, hipMemcpyHostToDevice);

  float *valor_anterior = (float *)malloc(6 * sizeof(float));

  #ifdef USE_MIDI
  /********************
   *  Inicializo MIDI *
   ********************/

  // Set the global sample rate before creating class instances.
  // Attempt to instantiate MIDI output class.
  RtMidiIn *midiin;
  try {
    midiin = new RtMidiIn();
  } catch (RtMidiError &error) {
    error.printMessage();
    exit(0);
  }

  midiin->openVirtualPort();
  vector<unsigned char> message(5);

  double stamp = midiin->getMessage(&message);
  int nbytes = message.size();

  Stk::setSampleRate(var->state()->getsamplerate());
  Stk::showWarnings(true);

  //  SineWave sine;
  RtWvOut *dac;
  try {
    // Define and open the default realtime output device for one-channel
    // playback
    dac = new RtWvOut(2);
  } catch (StkError &) {
    exit(1);
  }

  #endif

  #ifdef USE_USB
  /*********************************
  //        Configurar USB         *
  //********************************/
  AOAUsbConnectionManager connectionManager;
  connectionManager.setConnectionDetails(
    "Blastervla",
    "Guitar Simulator",
    "Guitar Simulator",
    "1.0",
    "http://d.hatena.ne.jp/thorikawa/",
    "0000000012345678",
    USB_IN, 
    USB_OUT, 
    USB_VID, 
    USB_PID, 
    USB_ACCESSORY_PID, 
    USB_ACCESSORY_PID_ALT
  );
  connectionManager.start(usbConnectionListener);
  #endif

  /*********************************
  // ACA EMPIEZA EL LOOP PRINCIPAL *
  //********************************/

  for (int i = 10; i < var->state()->getnFrames(); i++) {

    for (int ii = 0; ii < var->state()->getcantCuerdas(); ++ii) {
      tococ[ii] = 0;
    }

    #ifdef USE_MIDI
    nbytes = 1;
    while (nbytes > 0) {
      stamp = midiin->getMessage(&message);
      nbytes = message.size();

      if (nbytes > 0) {
        int tipoycanal = message[0];  // whatever;
        int tipo;                     //
        tipo =
            tipoycanal / 16;  // (message[0] & (1 << 2 | 1 << 3 | 1 << 4)) >> 2;
        int canal =
            tipoycanal %
            16;  //(message[0] & (1 << 5 | 1 << 6 | 1 << 7 | 1 << 8)) >> 2;

        if (debugMode) {
          printf(" Recibo el mensaje midi \n");
          printf(" # nbytes: %i \n", nbytes);
          printf(" # tipo: %i \n",tipo);
          printf(" # canal: %i \n", canal);
          printf(" # message 0 (tipoycanal): %i \n", tipoycanal);
          printf(" # message 1: %i \n", message[1]);
          printf(" # message 2: %i \n", message[2]);
          printf(" # message 3: %i \n", message[3]);
        }

        if (message[1] == 84 && tipo == 11) {
          retoque = true;
          printf("estoy retocando\n");
        }

        if (tipo == 9 && message[1] == 24 && debugMode) {
          imprimir_parametros(var->state()->getcantCuerdas(), var->state()->getcantMaximaNodos(),
                              var->state()->getmasaPorNodo(), var->state()->getfriccionSinDedo(),
                              Fext, mic, nodoscuer);
        }

        if (tipo == 9) {
          int cur = 5 - canal;     // 149-tipoycanal;
          int toque = message[2];  //-30;

          if (message[2] > 16 ) {
            caca[cur] = 1;
            freno[cur] = 0;
            if (!retoque){
		    tococ[cur] = 1;
                    intensidad = var->state()->getescalaIntensidad() * toque * toque * toque /
                         (60.0f + 120.0 * cur);
                    if (debugMode)
                   printf(
                     "Toco la cuerda, %i con la fuerza %i en %e , retoque = %i \n",
                         cur, message[2], var->state()->getcentro(), retoque);
                          una_gaussiana(Fext, var->state()->getcantMaximaNodos(), cur, intensidad,
                          var->state()->getcentro(), expo, nodoscuer);
                    hipMemcpy(d_Fext, Fext, mem_size_A, hipMemcpyHostToDevice);
                    fuerzavieja[cur] = message[2];
	    }
          } else if (message[2] == 0 || message[2] == 16) {
            tococ[cur] = 0;
            if (message[2] == 0 && !retoque) freno[cur] = 1;
             retoque = 0;
            if (debugMode) printf("Freno la cuerda %i \n", cur);
          }

          if (message[2] > 0 && cur >= 0 && cur <= 5) {
            double fdedo = 1 / 1.059463094;

            int value = 40;
            switch (cur) {
              case 0:
                value = 40;
                break;
              case 1:
                value = 45;
                break;
              case 2:
                value = 50;
                break;
              case 3:
                value = 55;
                break;
              case 4:
                value = 59;
                break;
              case 5:
                value = 64;
                break;
            }
            int fret = message[1] - value;
            dedo[cur] = (int)((float)nodoscuer[cur]) * pow(fdedo, fret);
          }
        }

        if (tipo == 14) {
          palanca = 1.0f + 0.002f * (message[2] - 64);
          if (debugMode) printf("Usa palanca \n ");
        }
        if (tipo == 11) {
          if (debugMode) printf("Canal controladores \n");

          if (message[1] == 74 || message[1] == 1) {
            var->state()->setmaxp(0.007 * message[2]);

            if (debugMode)
              printf(
                  " El maximo de la gaussiana de fricciones %e en la cuerda "
                  "%i\n",
                  var->state()->getmaxp(), cuerda_tocada);

            if (de_a_una) {
            	friccion_una_punta(var, fricc, dedo, cuerda_tocada);
            }
          }

          if (message[1] == 71 || message[1] == 2) {
            var->state()->setexpp(0.8f - 0.0060 * message[2]);

            if (debugMode)
              printf(" El exponente de la gaussiana de fricciones %e \n",
                     var->state()->getexpp());

            if (de_a_una) {
              friccion_una_punta(var, fricc, dedo, cuerda_tocada);
            }
          }

          if (message[1] == 91 || message[1] == 4) {
            for (int jk = 0; jk < var->state()->getcantCuerdas(); ++jk) {
              fricc[jk] = var->state()->getcuerdas(jk, "friccion") * message[2] *
                          0.03;  // 0.00000001;
            }

            if (debugMode) printf(" La friccion base es %e \n", fricc[0]);

            if (de_a_una) {
              friccion_una_punta(var, fricc, dedo, cuerda_tocada);
            }
          }

          if (message[1] == 93 || message[1] == 5) {
            expo = 0.00005 * (message[2] * message[2]) + 0.0005;
            if (debugMode)
              printf("El exponente para la fuerza es  %e \n", expo);

            gaussiana(Fext, var->state()->getcantMaximaNodos(), var->state()->getcantCuerdas(),
                      intensidad, var->state()->getcentro(), expo, nodoscuer);
            hipMemcpy(d_Fext, Fext, mem_size_A, hipMemcpyHostToDevice);
          }

          if (message[1] == 73 || message[1] == 8) {
            var->state()->setcentro(0.5 - ((float)message[2]) / 270.0);
            if (debugMode)
              printf("La fuerza se aplica en   %e \n", var->state()->getcentro());
            gaussiana(Fext, var->state()->getcantMaximaNodos(), var->state()->getcantCuerdas(),
                      intensidad, var->state()->getcentro(), expo, nodoscuer);
            hipMemcpy(d_Fext, Fext, mem_size_A, hipMemcpyHostToDevice);
          }

          if (message[1] == 72 || message[1] == 10) {
            var->state()->setescalaIntensidad(0.005 * message[2]);
            if (debugMode)
              printf("intbase es ahora   %e \n", var->state()->getescalaIntensidad());
            intensidad = var->state()->getescalaIntensidad() * 2000;
            gaussiana(Fext, var->state()->getcantMaximaNodos(), var->state()->getcantCuerdas(),
                      intensidad, var->state()->getcentro(), expo, nodoscuer);
            hipMemcpy(d_Fext, Fext, mem_size_A, hipMemcpyHostToDevice);
          }

          if (message[1] == 7) {
            volumen =
                (float)message[2] /
                ((float)30000.0);  // intensidad*(10-0.1)/(3.e-4 - 1.e-6) +
            if (debugMode) printf("El volumen es %e \n", volumen);
          }
        }

        if (canal == 48) {
          if (message[1] == 8) {
            cerr << " ************ Panico! reseteando todo ************" << endl;

            ceroInit(X, size_A);
            ceroInit(V, size_A);

            hipMemcpy(d_X, X, mem_size_A, hipMemcpyHostToDevice);
            hipMemcpy(d_V, V, mem_size_A, hipMemcpyHostToDevice);
          }

          if (message[1] == 7) {
          	imprimir = !imprimir;
            var->state()->setimprimir(imprimir);
            printf("¿estoy imprimiendo?  %s",
                   imprimir ? "Si!!!" : "NO!!!");
          }

          if (message[1] == 0) {
            d_pertu = d_pertup;
            printf("Ahora toco con pua \n");
          }

          if (message[1] == 1) {
            d_pertu = d_pertud;
            printf("Ahora toco con dedo \n");
          }

          if (message[1] == 3) {
            de_a_una = 0;
          }

          if (message[1] == 4 && debugMode) {
            imprimir_parametros(
                var->state()->getcantCuerdas(), var->state()->getcantMaximaNodos(),
                var->state()->getmasaPorNodo(), var->state()->getfriccionSinDedo(), Fext, mic,
                nodoscuer);
          }
        }

        if (message[1] == 64 && canal == 40) {
          pedal = message[2] == 127 ? 0 : (message[2] == 0 ? 1 : pedal);
          if (debugMode) cout << "Pedal " << (pedal ? "ON" : "OFF") << endl;
        }
      }
    }
    #endif

    #ifdef USE_USB
    initPresses();

    int response = 0;
    do {
        response = connectionManager.listenNow();
    } while (response == 0);

    deInitPresses();
    #endif

    hipMemcpy(d_tococ, tococ, bcur, hipMemcpyHostToDevice);
    hipMemcpy(d_freno, freno, bcur, hipMemcpyHostToDevice);
    hipMemcpy(d_dedo, dedo, var->state()->getcantCuerdas() * sizeof(int),
               hipMemcpyHostToDevice);

    if (var->state()->isChanged) {
      if (var->state()->isChangedmasaPorNodo()) {
          d_M = cudaFreeMallocAndCopy<float>(var->state()->getmasaPorNodo(), d_M);
          var->state()->setChangedmasaPorNodo();
      }

      if (var->state()->isChangedfriccionSinDedo()) {
        d_Fr = cudaFreeMallocAndCopy<float>(var->state()->getfriccionSinDedo(), d_Fr);
        var->state()->setChangedfriccionSinDedo();
      }

      if (var->state()->isChangedfriccionConDedo()) {
        d_Frd = cudaFreeMallocAndCopy<float>(var->state()->getfriccionConDedo(), d_Frd);
        var->state()->setChangedfriccionConDedo();
      }

      // Esperariamos que esto no cambie mucho (soft-realtime)
      if (var->state()->isChangedcuerdas() && (i % var->state()->getsoftrealtimeRefresh() == 0)) {
        for (int c=0; c < var->state()->getcantCuerdas(); ++c) {
            fricc[c] = var->state()->getcuerdas(c, "friccion");
        }
        var->state()->setChangedcuerdas();
      } 
      var->state()->isChanged = false;
    }

    // Imprimo variables de entrada

    avance<<<grid, threads>>>(
        d_X, d_V, d_Fext, d_salida, var->state()->getcantMaximaNodos(), d_M, d_Fr, d_Frd,
        var->state()->getdedoSize(), d_tococ, d_activa, d_freno, random, var->state()->getnbufferii(),
        d_dedo, d_dedov, d_pertu, d_mic, d_nodoscuer, pedal, var->state()->getcentro(),
        d_entrada, d_xmin, palanca);

    for (int kk = 0; kk < var->state()->getcantCuerdas(); kk++) {
      dedov[kk] = dedo[kk];
    }

    hipMemcpy(salida, d_salida, buffer_salida, hipMemcpyDeviceToHost);
    if (imprimir && i % 23 == 0) {
      hipMemcpy(X, d_X, mem_size_A, hipMemcpyDeviceToHost);
      hipMemcpy(dedo, d_dedo, var->state()->getcantCuerdas() * sizeof(int),
                 hipMemcpyDeviceToHost);
      fprintf(xyz, " %d \n", nodosreales + 2 * var->state()->getcantCuerdas());
      fprintf(xyz, "  \n");
      for (int ji = 0; ji < var->state()->getcantCuerdas(); ji++) {
        for (int jj = 0; jj < nodoscuer[ji]; jj++) {
          fprintf(xyz, " %d %E %E %E \n", ji + 6,
                  0.5 * X[jj + (ji)*var->state()->getcantMaximaNodos()], 1.2 * jj,
                  20.0 * ji);
        }
        fprintf(xyz, " %d %E %E %E \n", 26, 0.0, 1.2 * dedo[ji], 20.0 * ji);
        if (caca[ji] == 1) {
          caca[ji] = 0;
          fprintf(xyz, " %d %E %E %E \n", 1, 0.,
                  1.2 * var->state()->getcentro() * nodoscuer[ji], 20.0 * ji);

        } else {
          fprintf(xyz, " %d %E %E %E \n", 1, 50.,
                  1.2 * var->state()->getcentro() * nodoscuer[ji], 20.0 * ji);
        }
      }
    }
    StkFloat sal;
    StkFloat sal2;
    StkFrames frames(var->state()->getnbufferii(), 2);
    StkFloat *samples = &frames[0];

    unsigned int hop = frames.channels();

    for (uint jk = 0; jk < var->state()->getnbufferii(); jk++) {
      sal = 0;
      sal2 = 0;
      for (uint iii = 0; iii < var->state()->getcantCuerdas(); iii++) {
        sal2 += salida[jk + (2 * iii) * var->state()->getnbufferii()] * paneo[iii] *
                volcuer[iii];
        sal += salida[jk + (2 * iii + 1) * var->state()->getnbufferii()] *
               (1.0f - paneo[iii]) * volcuer[iii];
      }

      if (imprimir) {
        int sali = 20000 * sal2;
        int sali2 = 20000 * sal;
        fprintf(out, "  %d , %d \n", sali, sali2);
      }

      if (sal2 != sal2 || sal != sal) {
        ceroInit(X, size_A);
        ceroInit(V, size_A);

        hipMemcpy(d_X, X, mem_size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_V, V, mem_size_A, hipMemcpyHostToDevice);

        sal2 = 0.;
        sal = 0.;
      }

      *samples = sal2 * volumen;
      samples += 1;
      *samples = sal * volumen;
      samples += 1;
    }

    try {
      dac->tick(frames);
    } catch (StkError &) {
      goto cleanup;
    }
  }
  hipUnbindTexture(d_potencial_tex);
cleanup:

  return;
}

// ==================== Handling con USB ====================

typedef struct FingerPress {
    bool valid;
    bool isCejilla;
    int id;
    int node;
    int chord;
    int vertStretch;
    int pressure;
} FingerPress;

FingerPress* presses[MAX_FINGER_PRESS_AMOUNT]; // Andá a llegar a 10 taps, te reto...

void initPresses() {
    for (int i = 0; i < MAX_FINGER_PRESS_AMOUNT; i++) {
        presses[i] = malloc(sizeof(FingerPress));
    }
}

void deInitPresses() {
    for (int i = 0; i < MAX_FINGER_PRESS_AMOUNT; i++) {
        free(presses[i]);
    }
}

void agregarOActualizarPress(FingerPress *press) {
    // Encuentro posición libre en presses
    int positionForPress = -1;
    int freePosition = -1;
    for (uint8_t i = 0; i < MAX_FINGER_PRESS_AMOUNT && positionForPress == -1; i++) {
        if (presses[i]->id == press->id && presses[i]->valid) { // Ya lo tengo
            positionForPress = i;
        } else if (!presses[i]->valid && freePosition == -1) {
            // Me guardo la primera posición libre que encuentre, por si lo tengo que agregar
            freePosition = i;
        }
    }

    if (positionForPress == -1) { // Si no lo tenía
        positionForPress = freePosition; // Lo meto en un slot no válido
    }
    assert(positionForPress != -1);
    
    // Actualizo press
    presses[positionForPress]->valid = true;
    presses[positionForPress]->id = press->id;
    presses[positionForPress]->isCejilla = press->isCejilla;
    presses[positionForPress]->node = press->node;
    presses[positionForPress]->chord = press->chord;
    presses[positionForPress]->vertStretch = press->vertStretch;
    presses[positionForPress]->pressure = press->pressure;

    free(press);
}

void clearPresses() {
    for (uint8_t i = 0; i < MAX_FINGER_PRESS_AMOUNT; i++) {
        presses[i]->valid = false;
    }
}

void quitarPress(int id) {
    // Encuentro posición libre en presses
    bool eliminado;
    for (uint8_t i = 0; i < MAX_FINGER_PRESS_AMOUNT && !eliminado; i++) {
        if (presses[i]->id == id) { // Lo elimino
            presses[i]->valid = false;
            eliminado = true;
        }
    }
}

// ===== UTILS ======
void clearScreen()
{
    system("clear");
}
// ==================

void logPressess() {
    if (!debugMode) {
        clearScreen();
    }

    printf("Se están apretando los nodos: ============================\n");
    for (uint8_t i = 0; i < MAX_FINGER_PRESS_AMOUNT; i++) {
        if (presses[i]->valid) { // Lo elimino
            if (presses[i]->isCejilla) {
                printf("[ C ]");
            } else {
                printf("[ ");
                switch (presses[i]->chord) {
                    case 0:
                        printf("e ]");
                        break;
                    case 1:
                        printf("b ]");
                        break;
                    case 2:
                        printf("g ]");
                        break;
                    case 3:
                        printf("d ]");
                        break;
                    case 4:
                        printf("a ]");
                        break;
                    case 5:
                        printf("E ]");
                        break;
                }
            }
            printf("Node:  %i  |  VertStretch:  %i  |  Pressure:  %i\n", presses[i]->node, presses[i]->vertStretch, presses[i]->pressure);
        }
    }
    printf("==========================================================\n\n\n\n");
}

void usbConnectionListener(unsigned char* &dataBuff, int size) {
    clearPresses(); // Comentar si vamos por la alternativa de delete no implícito

    if (debugMode) {
        printf(" Se recibió mensaje USB \n");
        printf(" # Bytes: %i \n", size);
    }

    int intOffset = 0;
    // Nos movemos de a 5 bytes, que es el mínimo tamaño del paquete.
    for (offset = 0; offset < size; offset += 8) {
        int *buff = (int *) dataBuff;
        int op = buff[intOffset];

        if (debugMode) {
            printf(" # OP: %i\n", op);
        }
        
        switch (op) {
            case 0x2: { // Finger press
                FingerPress *press = malloc(sizeof(FingerPress));
                press->isCejilla = buff[intOffset + 1];
                press->node = buff[intOffset + 2];
                press->chord = buff[intOffset + 3];
                press->id = buff[intOffset + 4];
                press->vertStretch = buff[intOffset + 5];
                press->pressure = buff[intOffset + 6];

                agregarOActualizarPress(press);

                // Este es un paquete más grande de lo común
                // Lo aumentamos en la diferencia con el paquete más pequeño
                // 28 - 8 = 20
                intOffset += 7;
                offset += 20;

                break;
            }
            // Variante con finger release explícito
            /* 
            case 0x3: { // Finger release
                int id = buff[intOffset + 1];


                quitarPress(id);
                intOffset += 2;

                break;
            }
            */
            default: { // assume 8 byte packet
                intOffset += 2;
                break;
            }
        }
    }

    logPressess();
    actualizarEstado();
}

void actualizarEstado() {
    if (debugMode) {
        printf(" Recibo el mensaje midi \n");
        printf(" # nbytes: %i \n", nbytes);
        printf(" # tipo: %i \n",tipo);
        printf(" # canal: %i \n", canal);
        printf(" # message 0 (tipoycanal): %i \n", tipoycanal);
        printf(" # message 1: %i \n", message[1]);
        printf(" # message 2: %i \n", message[2]);
        printf(" # message 3: %i \n", message[3]);
    }

    // TODO: Pasar data con cada press!!
    for (uint8_t i = 0; i < MAX_FINGER_PRESS_AMOUNT; i++) {
        if (presses[i]->valid) {
            // Actualizo estado del sistema con la data de este press!
        }
    }
}